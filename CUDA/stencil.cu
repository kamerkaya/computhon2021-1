#include "hip/hip_runtime.h"
#include <stdio.h>
#include <omp.h>
#include "common.h"

#define RADIUS        512
#define BLOCK_SIZE    512
#define NUM_ELEMENTS  (1 << 24)

#define DEBUG
inline hipError_t checkCuda(hipError_t result) {
#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
  }
  #endif
  return result;
}

__global__ void stencil_1d_device(int *in, int *out, int N) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if(index < N) {
    // Apply the stencil
    int result = 0;
    for (int i = index ; i < index + 2*RADIUS + 1; i++)
      result += in[i];
    
    // Store the result
    out[index] = result;
  }
}

__global__ void stencil_1d_shared(int *in, int *out, int N) {
  __shared__ int temp[BLOCK_SIZE + 2 * RADIUS];
  int gindex = threadIdx.x + (blockIdx.x * blockDim.x) + RADIUS;
  if(gindex < N + RADIUS) {
    int lindex = threadIdx.x + RADIUS;
    
    // Read input elements into shared memory
    temp[lindex] = in[gindex];
    if (threadIdx.x < RADIUS) {
      temp[lindex - RADIUS] = in[gindex - RADIUS];
      temp[lindex + BLOCK_SIZE] = in[gindex + BLOCK_SIZE];
    }	
    
    // Make sure all threads get to this point before proceeding!
    __syncthreads();
    
    // Apply the stencil
    int result = 0;
    for (int offset = -RADIUS ; offset <= RADIUS ; offset++)
      result += temp[lindex + offset];
    
    // Store the result
    out[gindex - RADIUS] = result;
  }
}

int main() {
  hipSetDevice(0);

  unsigned int i;
  int* h_in = new int[NUM_ELEMENTS + 2 * RADIUS];
  int* h_out = new int[NUM_ELEMENTS];
  int *d_in, *d_out;
  
  // Initialize host data
  for( i = 0; i < (NUM_ELEMENTS + 2*RADIUS); ++i )
    h_in[i] = 1; // With a value of 1 and RADIUS of 3, all output values should be 7
  
  // Allocate space on the device
  cudaCheck( hipMalloc( &d_in, (NUM_ELEMENTS + 2*RADIUS) * sizeof(int)) );
  cudaCheck( hipMalloc( &d_out, NUM_ELEMENTS * sizeof(int)) );
  
  //Timing structures
  hipEvent_t start,stop;
  float elapsedTime;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  
  // Copy input data to device
  cudaCheck( hipMemcpy( d_in, h_in, (NUM_ELEMENTS + 2*RADIUS) * sizeof(int), hipMemcpyHostToDevice) );
  
  hipEventRecord(start,0);
  cudaCheck( hipMemcpy( d_in, h_in, (NUM_ELEMENTS + 2*RADIUS) * sizeof(int), hipMemcpyHostToDevice) );
  stencil_1d_shared<<< (NUM_ELEMENTS + BLOCK_SIZE - 1)/BLOCK_SIZE, BLOCK_SIZE >>> (d_in, d_out, NUM_ELEMENTS);
  cudaCheck( hipMemcpy( h_out, d_out, NUM_ELEMENTS * sizeof(int), hipMemcpyDeviceToHost) );
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  checkCuda(hipGetLastError());
  hipEventElapsedTime(&elapsedTime,start,stop);
  
  // Verify every out value is 2*RADIUS + 1
  for( i = 0; i < NUM_ELEMENTS; ++i ) {
    if (h_out[i] != 2 * RADIUS + 1) {
      printf("Element h_out[%d] == %d != %d\n", i, h_out[i], 2*RADIUS+1);
      break;
    }
  }
  if (i == NUM_ELEMENTS) printf("SUCCESS GPU_SHARED in %f mseconds!\n", elapsedTime);
  
  hipEventRecord(start,0);
  cudaCheck( hipMemcpy( d_in, h_in, (NUM_ELEMENTS + 2*RADIUS) * sizeof(int), hipMemcpyHostToDevice) );
  stencil_1d_device<<< (NUM_ELEMENTS + BLOCK_SIZE - 1)/BLOCK_SIZE, BLOCK_SIZE >>> (d_in, d_out, NUM_ELEMENTS);
  cudaCheck( hipMemcpy( h_out, d_out, NUM_ELEMENTS * sizeof(int), hipMemcpyDeviceToHost) );
  checkCuda(hipGetLastError());
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  
  hipEventElapsedTime(&elapsedTime,start,stop);
  // Verify every out value is 2*RADIUS + 1
  for( i = 0; i < NUM_ELEMENTS; ++i ) {
    if (h_out[i] != 2 * RADIUS + 1) {
      printf("Element h_out[%d] == %d != %d\n", i, h_out[i], 2*RADIUS+1);
      break;
    }
  }
  if (i == NUM_ELEMENTS) printf("SUCCESS GPU_DEVICE in %f mseconds!\n", elapsedTime);
  
  double startt = omp_get_wtime();
#pragma omp parallel for schedule(static)
  for(int i = 0; i < NUM_ELEMENTS; i++) {
    int sum = 0;
    for(int j = i; j < i + 2*RADIUS + 1; j++) {
      sum += h_in[j];
    }
    h_out[i] = sum;
  }
  double endt = omp_get_wtime();
  
  for( i = 0; i < NUM_ELEMENTS; ++i ) {
    if (h_out[i] != 2 * RADIUS + 1) {
      printf("Element h_out[%d] == %d != %d\n", i, h_out[i], 2*RADIUS+1);
      break;
    }
  }
  if (i == NUM_ELEMENTS) printf("SUCCESS CPU in %f mseconds!\n", 1000 * (endt - startt));
  // Free out memory
  hipFree(d_in);
  hipFree(d_out);
  
  return 0;
}

